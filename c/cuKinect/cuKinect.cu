#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cutil_inline.h>

#include "libfreenect.h"

uint16_t t_gamma[2048];
uint8_t* gl_depth[2];

void depth_cb(freenect_device *dev, freenect_depth *depth, uint32_t timestamp)
{
    printf("called depth_cb()\n");
	int i;

	for (i=0; i<FREENECT_FRAME_PIX; i++) {
		int pval = t_gamma[depth[i]];
		int lb = pval & 0xff;
		switch (pval>>8) {
			case 0:
				gl_depth[0][3*i+0] = 255;
				gl_depth[0][3*i+1] = 255-lb;
				gl_depth[0][3*i+2] = 255-lb;
				break;
			case 1:
				gl_depth[0][3*i+0] = 255;
				gl_depth[0][3*i+1] = lb;
				gl_depth[0][3*i+2] = 0;
				break;
			case 2:
				gl_depth[0][3*i+0] = 255-lb;
				gl_depth[0][3*i+1] = 255;
				gl_depth[0][3*i+2] = 0;
				break;
			case 3:
				gl_depth[0][3*i+0] = 0;
				gl_depth[0][3*i+1] = 255;
				gl_depth[0][3*i+2] = lb;
				break;
			case 4:
				gl_depth[0][3*i+0] = 0;
				gl_depth[0][3*i+1] = 255-lb;
				gl_depth[0][3*i+2] = 255;
				break;
			case 5:
				gl_depth[0][3*i+0] = 0;
				gl_depth[0][3*i+1] = 0;
				gl_depth[0][3*i+2] = 255-lb;
				break;
			default:
				gl_depth[0][3*i+0] = 0;
				gl_depth[0][3*i+1] = 0;
				gl_depth[0][3*i+2] = 0;
				break;
		}
	}
}

int main(int argc, char** argv) 
{
    int devID = 0;
    hipDeviceProp_t deviceProps;

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if (cutCheckCmdLineFlag(argc, (const char**)argv, "device")) {
        cutilDeviceInit(argc, argv);
    } else {
        devID = cutGetMaxGflopsDeviceId();
        cutilSafeCall(hipSetDevice(devID));
    }

    // get number of SMs on this GPU
    cutilSafeCall(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s] has %d Multi-Processors\n", 
           deviceProps.name, deviceProps.multiProcessorCount);
    
    
    freenect_context *f_ctx;
    freenect_device *f_dev;
	  
	if (freenect_init(&f_ctx, NULL) < 0) {
        printf("freenect_init() failed\n");
		exit(0);
	}

	int nr_devices = freenect_num_devices (f_ctx);
	printf ("Number of Kinect devices found: %d\n", nr_devices);

	if (nr_devices < 1)
		exit(0);

	if (freenect_open_device(f_ctx, &f_dev, 0) < 0) {
		printf("Could not open Kinect device\n");
		exit(0);
	}
	
	int i;
	for (i=0; i<2048; i++) {
		float v = i/2048.0;
		v = powf(v, 3)* 6;
		t_gamma[i] = v*6*256;
	}
	
    // allocate pinned buffers
	hipHostAlloc(&(gl_depth[0]), sizeof(uint8_t)*640*480*4, 0);
	hipHostAlloc(&(gl_depth[1]), sizeof(uint8_t)*640*480*4, 0);
	
	freenect_set_depth_callback(f_dev, depth_cb);
	freenect_set_depth_format(f_dev, FREENECT_FORMAT_11_BIT);
    
    freenect_start_depth(f_dev);
    
    int bufNum = 0;
    void *pGPUbuf[2];
    
    // allocate buffers
    hipMalloc(&(pGPUbuf[0]), sizeof(uint8_t)*640*480*4);
    hipMalloc(&(pGPUbuf[1]), sizeof(uint8_t)*640*480*4);
    
    // inf. loop
    while (freenect_process_events(f_ctx) >= 0) {
        printf("copy gl_depth[%d] to pGPUbuf[%d]\n", (bufNum+1)%2, bufNum);
        hipMemcpyAsync(pGPUbuf[bufNum], gl_depth[(bufNum+1)%2], sizeof(uint8_t)*640*480*4, hipMemcpyHostToDevice, 0);
        
        //kernel call here
        
        // get next frame... 
        printf("copy gl_depth[0] to gl_depth[%d]\n", bufNum);
        hipMemcpy(gl_depth[bufNum], gl_depth[0], sizeof(uint8_t)*640*480*4, hipMemcpyHostToHost);
        
        hipDeviceSynchronize();
        bufNum++; bufNum %=2;
    }
    
    // free buffers
    hipHostFree(gl_depth[0]);
    hipHostFree(gl_depth[1]);
    hipFree(pGPUbuf[0]);
    hipFree(pGPUbuf[1]);
    
    hipDeviceReset();
    cutilExit(argc, argv);
    return 0;
}
